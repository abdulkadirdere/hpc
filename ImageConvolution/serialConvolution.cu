#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <vector>
#include <math.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "helper/inc/helper_functions.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include "helper/inc/hip/hip_runtime_api.h" // helper functions for CUDA error check

#define length(arr) ((int) (sizeof (arr) / sizeof (arr)[0]))

const int size=3;
const int mask_size = 3;
const int offset = floor(mask_size/2);
const int output_size = size + 2*offset;

const double mask[3][3] = {
    {1, 1, 1},
    {1, 1, 1},
    {1, 1, 1},
};

// const int mask[5][5] = {
//     {1, 1, 1, 1, 1},
//     {1, 1, 1, 1, 1},
//     {1, 1, 1, 1, 1},
//     {1, 1, 1, 1, 1},
//     {1, 1, 1, 1, 1},
// };

// const int averaging[3][3] = {
//     {2, 2, 2},
//     {2, 2, 2},
//     {2, 2, 2},
// };


void printArray(double **array, int r, int c) {
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++) {
            printf("%.0f ", array[i][j]);
        }
        printf("\n");
    }
}

int randomNumberGeneration(int upperBound, int lowerBound) {
    int num = (rand() % (upperBound - lowerBound + 1)) + lowerBound;
    return num;
}

double **createMatrix(int m, int n) {
    double **array = (double **)malloc(m * sizeof(double *));
    for (int i = 0; i < m; i++) {
        array[i] = (double *)malloc(n * sizeof(double));
    }

    return array;
}

double **createData(double **array, int size, int dimension) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < dimension; j++) {
            array[i][j] = randomNumberGeneration(9, 1);
        }
    }
    return array;
}

double **padArray(double **input, double **output) {
    int range = output_size - offset;
    // printf("%d \n", range);

    // pad the array
    for (int i = offset; i < range; i++) {
        for (int j = offset; j < range; j++) {
            output[i][j] = input[i-offset][j-offset];
        }
    }
    return output;
}

double **unpad(double **input, double **output) {
    int range = output_size - offset;

    // unpad the array
    for (int i = 0; i < range; i++) {
        for (int j = 0; j < range; j++) {
            output[i][j] = input[i+offset][j+offset];
        }
    }
    return output;
}


double applyMask(double **array, int row, int col){
    int n_size = offset * 2 + 1;

    // neighbours of giving location
    double **neighbours = createMatrix(n_size, n_size);

    int range = output_size - offset;
    // for (int i=row; i < range; i++){
    //     for(int j=col; j < range; j++){
    //         neighbours[row-offset][col-offset] = array[row-offset][col-offset];
    //     }
    // }
    // printArray(neighbours, n_size, n_size);

    neighbours[0][0] = array[row-1][col-1]; // top_left
    neighbours[0][1] = array[row-1][col]; // top_middle
    neighbours[0][2] = array[row-1][col+1]; //top_right

    neighbours[1][0] = array[row][col-1]; //middle_left
    neighbours[1][1] = array[row][col]; //middle_middle
    neighbours[1][2] = array[row][col+1]; //middle_right

    neighbours[2][0] = array[row+1][col-1]; //bottom_left
    neighbours[2][1] = array[row+1][col]; //bottom_middle
    neighbours[2][2] = array[row+1][col+1]; //bottom_right


    double **convolution = createMatrix(n_size, n_size);
    int value = 0;

    for (int r=0; r<3; r++){
        for(int c=0; c<3; c++){
            convolution[r][c] = mask[r][c] * neighbours[r][c];
            value = value + convolution[r][c];
        }
    }
    // printf("%d \n", value);
    // printArray(convolution, offset, offset);

    return value;
}

double **serial_convolution(double **input, double **output){
    int range = output_size - offset;
    // printf("%d ", range);

    for (int i = offset; i<range; i++){
        for (int j = offset; j<range; j++){
            output[i][j] = applyMask(input, i, j);
        }
    }
    return output;
}


int main(int argc, char **argv){
    int devID = findCudaDevice(0, 0);
    hipGetDeviceProperties(0, 0);

    const char *imageFilename = "lena_bw.pgm";

    // load image from disk
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);


    // printArray(hData, width, width);


    // double **input = createMatrix(size,size);
    // double **padded = createMatrix(output_size, output_size);
    // double **output = createMatrix(output_size, output_size);
    // double **unpadded = createMatrix(output_size, output_size);

    // input = createData(input, size, size);
    // // printArray(input, size, size);
    // printf("offset size: %d \n", offset);

    // // pad the given array
    padded = padArray(hData, padded);

    // printArray(padded, output_size, output_size);
    // printf("padded output \n");

    // output = serial_convolution(padded, output);
    // printArray(output, output_size, output_size);

    // unpadded = unpad(output, unpadded);
    // printf("unpadded output \n");
    // printArray(unpadded, size, size);

}